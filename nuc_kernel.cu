#include "hip/hip_runtime.h"
#include "nuc_kernel.h"
#include <stdio.h>

#define THREADS 512

__global__ void nuc(uint16_t *out,const float *gain,const float *offset, int size) {
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i<size)	out[i] = (uint16_t)(gain[i]*(out[i] - offset[i]));
}


void GPUnuc(uint16_t *out, float *gain, float *offset, int n){
    // d_offset zero-copy
    float *d_offset = NULL;
    hipHostRegister(offset, n*sizeof(float), hipHostRegisterMapped);
    hipHostGetDevicePointer((void **)&d_offset, (void *)offset, 0);
    // d_gain zero-copy
    float *d_gain = NULL;
    hipHostRegister(gain, n*sizeof(float), hipHostRegisterMapped);
    hipHostGetDevicePointer((void **)&d_gain, (void *)gain, 0);
    // d_out zero-copy
    uint16_t *d_out = NULL;
    hipHostRegister(out, n*sizeof(uint16_t), hipHostRegisterMapped);
    hipHostGetDevicePointer((void **)&d_out, (void *)out, 0);

    int threadsPerBlock = THREADS;
    int blocksPerGrid =(n + threadsPerBlock - 1) / threadsPerBlock;

    nuc<<<blocksPerGrid,threadsPerBlock>>>(d_out, d_gain, d_offset,n);

    // clean up
    hipHostUnregister(d_out);
    hipHostUnregister(d_offset);
    hipHostUnregister(d_gain);

    return;
};
