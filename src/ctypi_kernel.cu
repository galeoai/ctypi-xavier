#include "hip/hip_runtime.h"
#define THREADS 512
#include <cstdint>
#include <stdio.h>

#include "ctypi_kernel.h"

#define KERNEL_RADIUS 3
#define KERNEL_LENGTH (2 * KERNEL_RADIUS + 1)

__constant__ float c1[KERNEL_LENGTH] = {
    0.0116850998497429921230139626686650444753468036651611328125,
    -0.0279730819380002923568717676516826031729578971862792968750,
    0.2239007887600356350166208585505955852568149566650390625000,
    0.5847743866564433234955799889576155692338943481445312500000,
    0.2239007887600356350166208585505955852568149566650390625000,
    -0.0279730819380002923568717676516826031729578971862792968750,
    0.0116850998497429921230139626686650444753468036651611328125 };

///////////////////////////////////////////////////////////////////////////////
//                                    diff                                   //
///////////////////////////////////////////////////////////////////////////////
// out = im1-im2 
__global__ void diff(uint16_t *out, uint16_t *im1,uint16_t *im2, int size){
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i<size) {
	out[i] = im1[i]-im2[i];
    };
};

void GPUdiff(uint16_t *out, uint16_t *im1,uint16_t *im2, int size){
    uint16_t *d_out = NULL;
    hipHostRegister(out, size*sizeof(uint16_t), hipHostRegisterMapped);
    hipHostGetDevicePointer((void **)&d_out, (void *)out, 0);
    uint16_t *d_im1 = NULL;
    hipHostRegister(im1, size*sizeof(uint16_t), hipHostRegisterMapped);
    hipHostGetDevicePointer((void **)&d_im1, (void *)im1, 0);
    uint16_t *d_im2 = NULL;
    hipHostRegister(im2, size*sizeof(uint16_t), hipHostRegisterMapped);
    hipHostGetDevicePointer((void **)&d_im2, (void *)im2, 0);

    int threadsPerBlock = THREADS;
    int blocksPerGrid =(size + threadsPerBlock - 1) / threadsPerBlock;

    diff<<<blocksPerGrid,threadsPerBlock>>>(d_out,d_im1,d_im2,size);
    //printf("calling the kernel\n");
    //hipDeviceSynchronize();
    //hipError_t cudaerr = hipDeviceSynchronize();
    //if (cudaerr != hipSuccess)
    //    printf("kernel launch failed with error \"%s\".\n",
    //           hipGetErrorString(cudaerr));
    
    //printf("done\n");
    // clean up
    hipHostUnregister(d_out);
    hipHostUnregister(d_im1);
    hipHostUnregister(d_im2);
}

///////////////////////////////////////////////////////////////////////////////
//                                  filter_x                                 //
///////////////////////////////////////////////////////////////////////////////
__global__ void filter_x(uint16_t *out,
			 uint16_t *in,
			 int imageW,
			 int imageH)
{
    int i0 = blockIdx.x*blockDim.x + threadIdx.x;
    int i1 = blockIdx.y*blockDim.y + threadIdx.y;

    int ii0 = threadIdx.x;
    // copy pixel value to shared memory
    __shared__ uint16_t s[THREADS + 2*KERNEL_RADIUS];
    s[KERNEL_RADIUS + ii0] = in[i0+imageH*i1];
    __syncthreads();

    #pragma unroll
    for (int j0 = -KERNEL_RADIUS; j0 < KERNEL_RADIUS; ++j0) {
	out[i0+imageH*i1] += s[KERNEL_RADIUS + ii0 + j0]*c1[KERNEL_RADIUS+j0];
    };

};


void GPUfilter_x(uint16_t *out, uint16_t *in,int imageW,int imageH){
    int size = imageW*imageH;
    uint16_t *d_out = NULL;
    hipHostRegister(out, size*sizeof(uint16_t), hipHostRegisterMapped);
    hipHostGetDevicePointer((void **)&d_out, (void *)out, 0);
    uint16_t *d_in = NULL;
    hipHostRegister(in, size*sizeof(uint16_t), hipHostRegisterMapped);
    hipHostGetDevicePointer((void **)&d_in, (void *)in, 0);

    //int threadsPerBlock = THREADS;
    //int blocksPerGrid =(size + threadsPerBlock - 1) / threadsPerBlock;

    dim3 threadsPerBlock(512,1);
    dim3 numBlocks(imageW/threadsPerBlock.x, imageH/threadsPerBlock.y);
    
    filter_x<<<numBlocks,threadsPerBlock>>>(d_out,d_in,imageW,imageH);
    //printf("calling the kernel\n");
    //hipDeviceSynchronize();
    //hipError_t cudaerr = hipDeviceSynchronize();
    //if (cudaerr != hipSuccess)
    //    printf("kernel launch failed with error \"%s\".\n",
    //           hipGetErrorString(cudaerr));
    
    //printf("done\n");
    // clean up
    hipHostUnregister(d_out);
    hipHostUnregister(d_in);
}

///////////////////////////////////////////////////////////////////////////////
//                                  filter_y                                 //
///////////////////////////////////////////////////////////////////////////////
__global__ void filter_y(uint16_t *out,
			 uint16_t *in,
			 int imageW,
			 int imageH)
{
    int i0 = blockIdx.x*blockDim.x + threadIdx.x;
    int i1 = blockIdx.y*blockDim.y + threadIdx.y;

    int ii0 = threadIdx.x;
    int ii1 = threadIdx.y;
    // copy pixel value to shared memory
    __shared__ uint16_t s[32][16 + 2*KERNEL_RADIUS];
    s[ii0][ii1+KERNEL_RADIUS] = in[i0+imageW*i1];
    // lower
    if(ii1==0) s[ii0][0] = in[i0+imageW*max(i1-3,0)];
    if(ii1==1) s[ii0][1] = in[i0+imageW*max(i1-3,0)];
    if(ii1==2) s[ii0][2] = in[i0+imageW*max(i1-3,0)];
    // upper
    if(ii1==13) s[ii0][KERNEL_RADIUS+16+0] = in[i0+imageW*min(i1+3,imageH)];
    if(ii1==14) s[ii0][KERNEL_RADIUS+16+1] = in[i0+imageW*min(i1+3,imageH)];
    if(ii1==15) s[ii0][KERNEL_RADIUS+16+2] = in[i0+imageW*min(i1+3,imageH)];

    __syncthreads();

    #pragma unroll
    for (int j1 = -KERNEL_RADIUS; j1 < KERNEL_RADIUS; ++j1) {
	out[i0+imageW*i1] += s[ii0][ii1+KERNEL_RADIUS+j1] *
	    c1[KERNEL_RADIUS+j1];
    };
};


void GPUfilter_y(uint16_t *out, uint16_t *in, int imageW, int imageH){
    int size = imageW*imageH;
    uint16_t *d_out = NULL;
    hipHostRegister(out, size*sizeof(uint16_t), hipHostRegisterMapped);
    hipHostGetDevicePointer((void **)&d_out, (void *)out, 0);
    uint16_t *d_in = NULL;
    hipHostRegister(in, size*sizeof(uint16_t), hipHostRegisterMapped);
    hipHostGetDevicePointer((void **)&d_in, (void *)in, 0);

    dim3 threadsPerBlock(32,16);
    dim3 numBlocks(imageW/threadsPerBlock.x, imageH/threadsPerBlock.y);
    
    filter_y<<<numBlocks,threadsPerBlock>>>(d_out,d_in,imageW,imageH);
    // clean up
    hipHostUnregister(d_out);
    hipHostUnregister(d_in);
};


///////////////////////////////////////////////////////////////////////////////
//                                    grad                                   //
///////////////////////////////////////////////////////////////////////////////

__global__ void grad(int *px, int *py,
		     uint16_t *im,  int imageW, int imageH){
    int i0 = blockIdx.x*blockDim.x + threadIdx.x;
    int i1 = blockIdx.y*blockDim.y + threadIdx.y;

    int ii0 = threadIdx.x;
    int ii1 = threadIdx.y;
    // copy pixel value to shared memory
    __shared__ uint16_t s[32 + 2][16 + 2];
    s[ii0+1][ii1+1] = im[i0+imageW*i1];

    if(ii0==0)	s[0][ii1+1]  = im[i0-1+imageW*i1];
    if(ii0==31) s[33][ii1+1] = im[i0+1+imageW*i1];
    if(ii1==0)	s[ii0+1][0]  = im[i0+imageW*(i1-1)];
    if(ii1==15) s[ii0+1][17] = im[i0+imageW*(i1+1)];

    __syncthreads();

    px[i0+imageW*i1] = (s[ii0+1+1][ii1+1] - s[ii0+1-1][ii1+1])/2;
    py[i0+imageW*i1] = (s[ii0+1][ii1+1+1] - s[ii0+1][ii1+1-1])/2;
};

void GPUgrad(int *px, int *py, uint16_t *im, int imageW, int imageH){
    int size = imageW*imageH;
    int *d_px = NULL;
    hipHostRegister(px, size*sizeof(int), hipHostRegisterMapped);
    hipHostGetDevicePointer((void **)&d_px, (void *)px, 0);
    int *d_py = NULL;
    hipHostRegister(py, size*sizeof(int), hipHostRegisterMapped);
    hipHostGetDevicePointer((void **)&d_py, (void *)py, 0);
    uint16_t *d_im = NULL;
    hipHostRegister(im, size*sizeof(uint16_t), hipHostRegisterMapped);
    hipHostGetDevicePointer((void **)&d_im, (void *)im, 0);

    dim3 threadsPerBlock(32,16);
    dim3 numBlocks(imageW/threadsPerBlock.x, imageH/threadsPerBlock.y);

    grad<<<numBlocks,threadsPerBlock>>>(d_px, d_py, d_im, imageW, imageH);

    // clean up
    hipHostUnregister(d_px);
    hipHostUnregister(d_py);
    hipHostUnregister(d_im);
};

///////////////////////////////////////////////////////////////////////////////
//                                    sum                                    //
///////////////////////////////////////////////////////////////////////////////

__device__ void warpReduce(volatile int *sdata, unsigned int tid) {
    sdata[tid] += sdata[tid + 32];
    sdata[tid] += sdata[tid + 16];
    sdata[tid] += sdata[tid +  8];
    sdata[tid] += sdata[tid +  4];
    sdata[tid] += sdata[tid +  2];
    sdata[tid] += sdata[tid +  1];
};

__global__ void sum(int *res, int *im,  int size){
    __shared__ int sdata[512];
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*(512*2) + tid;
    unsigned int gridSize =  512*2*gridDim.x;
    sdata[tid] = 0;
    while (i < size) {
	sdata[tid] += im[i] + im[i+512];
	i += gridSize;
    }
    __syncthreads();
    if (tid < 256) { sdata[tid] += sdata[tid + 256]; } __syncthreads();
    if (tid < 128) { sdata[tid] += sdata[tid + 128]; } __syncthreads();
    if (tid <  64) { sdata[tid] += sdata[tid +	64]; } __syncthreads();
    
    if (tid < 32) warpReduce(sdata, tid);
    
    if (tid == 0) res[blockIdx.x] = sdata[0];
};

int GPUsum(int *im,  int size){
    int *d_im = NULL;
    hipHostRegister(im, size*sizeof(int), hipHostRegisterMapped);
    hipHostGetDevicePointer((void **)&d_im, (void *)im, 0);
    
    int threadsPerBlock = THREADS;
    int blocksPerGrid =(size + threadsPerBlock - 1) / threadsPerBlock;

    int *d_tmp;
    hipMalloc((void **)&d_tmp,blocksPerGrid*sizeof(int));
    int *d_res;
    hipMalloc((void **)&d_res,1*sizeof(int));
    int res = INT_MIN; //init with "error" number 
    
    sum<<<blocksPerGrid, threadsPerBlock>>>(d_tmp, d_im, size);
    sum<<<1, threadsPerBlock>>>(d_res, d_tmp, blocksPerGrid);
    
    hipDeviceSynchronize();
    // copy results back to the cpu
    hipMemcpy(&res, d_res, 1*sizeof(int), hipMemcpyDeviceToHost);
    // clean up
    hipHostUnregister(d_im);
    hipHostUnregister(d_tmp);
    hipHostUnregister(d_res);
    
    return res;
};

///////////////////////////////////////////////////////////////////////////////
//                                    dot                                    //
///////////////////////////////////////////////////////////////////////////////
__global__ void dot(int *res, int *im1, int *im2, int size){
    __shared__ int sdata[512];
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*(512*2) + tid;
    unsigned int gridSize =  512*2*gridDim.x;
    sdata[tid] = 0;
    while (i < size) {
	sdata[tid] += im1[i]*im2[i] + im1[i+512]*im2[i+512];
	i += gridSize;
    }
    __syncthreads();
    if (tid < 256) { sdata[tid] += sdata[tid + 256]; } __syncthreads();
    if (tid < 128) { sdata[tid] += sdata[tid + 128]; } __syncthreads();
    if (tid <  64) { sdata[tid] += sdata[tid +	64]; } __syncthreads();
    
    if (tid < 32) warpReduce(sdata, tid);
    
    if (tid == 0) res[blockIdx.x] = sdata[0];
};

int GPUdot(int *im1, int *im2,  int size){
    int *d_im1 = NULL;
    hipHostRegister(im1, size*sizeof(int), hipHostRegisterMapped);
    hipHostGetDevicePointer((void **)&d_im1, (void *)im1, 0);
    int *d_im2 = NULL;
    hipHostRegister(im2, size*sizeof(int), hipHostRegisterMapped);
    hipHostGetDevicePointer((void **)&d_im2, (void *)im2, 0);
    
    int threadsPerBlock = THREADS;
    int blocksPerGrid =(size + threadsPerBlock - 1) / threadsPerBlock;

    int *d_tmp;
    hipMalloc((void **)&d_tmp,blocksPerGrid*sizeof(int));
    int *d_res;
    hipMalloc((void **)&d_res,1*sizeof(int));
    int res = INT_MIN; //init with "error" number 
    
    dot<<<blocksPerGrid, threadsPerBlock>>>(d_tmp, d_im1, d_im2, size);
    sum<<<1, threadsPerBlock>>>(d_res, d_tmp, blocksPerGrid);
    
    hipDeviceSynchronize();
    // copy results back to the cpu
    hipMemcpy(&res, d_res, 1*sizeof(int), hipMemcpyDeviceToHost);
    // clean up
    hipHostUnregister(d_im1);
    hipHostUnregister(d_im2);
    hipHostUnregister(d_tmp);
    hipHostUnregister(d_res);
    
    return res;
};
