#include "hip/hip_runtime.h"
#define THREADS 512
#include <cstdint>
#include <stdio.h>

#include "ctypi_kernel.h"

#define KERNEL_RADIUS 3
#define KERNEL_LENGTH (2 * KERNEL_RADIUS + 1)

__constant__ float c1[KERNEL_LENGTH] = {
    0.0116850998497429921230139626686650444753468036651611328125,
    -0.0279730819380002923568717676516826031729578971862792968750,
    0.2239007887600356350166208585505955852568149566650390625000,
    0.5847743866564433234955799889576155692338943481445312500000,
    0.2239007887600356350166208585505955852568149566650390625000,
    -0.0279730819380002923568717676516826031729578971862792968750,
    0.0116850998497429921230139626686650444753468036651611328125 };

///////////////////////////////////////////////////////////////////////////////
//                                    diff                                   //
///////////////////////////////////////////////////////////////////////////////
// out = im1-im2 
__global__ void diff(uint16_t *out, uint16_t *im1,uint16_t *im2, int size){
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i<size) {
	out[i] = im1[i]-im2[i];
    };
};

void GPUdiff(uint16_t *out, uint16_t *im1,uint16_t *im2, int size){
    uint16_t *d_out = NULL;
    hipHostRegister(out, size*sizeof(uint16_t), hipHostRegisterMapped);
    hipHostGetDevicePointer((void **)&d_out, (void *)out, 0);
    uint16_t *d_im1 = NULL;
    hipHostRegister(im1, size*sizeof(uint16_t), hipHostRegisterMapped);
    hipHostGetDevicePointer((void **)&d_im1, (void *)im1, 0);
    uint16_t *d_im2 = NULL;
    hipHostRegister(im2, size*sizeof(uint16_t), hipHostRegisterMapped);
    hipHostGetDevicePointer((void **)&d_im2, (void *)im2, 0);

    int threadsPerBlock = THREADS;
    int blocksPerGrid =(size + threadsPerBlock - 1) / threadsPerBlock;

    diff<<<blocksPerGrid,threadsPerBlock>>>(d_out,d_im1,d_im2,size);
    //printf("calling the kernel\n");
    //hipDeviceSynchronize();
    //hipError_t cudaerr = hipDeviceSynchronize();
    //if (cudaerr != hipSuccess)
    //    printf("kernel launch failed with error \"%s\".\n",
    //           hipGetErrorString(cudaerr));
    
    //printf("done\n");
    // clean up
    hipHostUnregister(d_out);
    hipHostUnregister(d_im1);
    hipHostUnregister(d_im2);
}

///////////////////////////////////////////////////////////////////////////////
//                                  filter_x                                 //
///////////////////////////////////////////////////////////////////////////////
__global__ void filter_x(uint16_t *out,
			 uint16_t *in,
			 int imageW,
			 int imageH)
{
    int i0 = blockIdx.x*blockDim.x + threadIdx.x;
    int i1 = blockIdx.y*blockDim.y + threadIdx.y;

    //copy line to shared memory
    //int  ind = threadIdx.x;
    //__shared__ uint16_t tmp[THREADS];
    //tmp[ind] = in[i0+imageH*i1];
    //__syncthreads();
    // 
    //for (int j0 = -KERNEL_RADIUS; j0 < KERNEL_RADIUS; ++j0) {
    // 	if( ((ind+j0)>=0) && ((ind+j0)<)) {
    // 	    out[i0+imageH*i1]+=tmp[ind+j0]*c1[KERNEL_RADIUS+j0];
    // 	};
    //};
    #pragma unroll
    for (int j0 = -KERNEL_RADIUS; j0 < KERNEL_RADIUS; ++j0) {
	if( ((i0+j0)>=0) && ((i0+j0)<imageW)) {
	    out[i0+imageH*i1] += in[i0+imageH*i1+j0]*c1[KERNEL_RADIUS+j0];
	};
    };

};


void GPUfilter_x(uint16_t *out, uint16_t *in,int imageW,int imageH){
    int size = imageW*imageH;
    uint16_t *d_out = NULL;
    hipHostRegister(out, size*sizeof(uint16_t), hipHostRegisterMapped);
    hipHostGetDevicePointer((void **)&d_out, (void *)out, 0);
    uint16_t *d_in = NULL;
    hipHostRegister(in, size*sizeof(uint16_t), hipHostRegisterMapped);
    hipHostGetDevicePointer((void **)&d_in, (void *)in, 0);

    //int threadsPerBlock = THREADS;
    //int blocksPerGrid =(size + threadsPerBlock - 1) / threadsPerBlock;

    dim3 threadsPerBlock(512,1);
    dim3 numBlocks(imageW/threadsPerBlock.x, imageH/threadsPerBlock.y);
    
    filter_x<<<numBlocks,threadsPerBlock>>>(d_out,d_in,imageW,imageH);
    //printf("calling the kernel\n");
    //hipDeviceSynchronize();
    //hipError_t cudaerr = hipDeviceSynchronize();
    //if (cudaerr != hipSuccess)
    //    printf("kernel launch failed with error \"%s\".\n",
    //           hipGetErrorString(cudaerr));
    
    //printf("done\n");
    // clean up
    hipHostUnregister(d_out);
    hipHostUnregister(d_in);
}

///////////////////////////////////////////////////////////////////////////////
//                                  filter_y                                 //
///////////////////////////////////////////////////////////////////////////////
__global__ void filter_y(uint16_t *out,
			 uint16_t *in,
			 int imageW,
			 int imageH)
{
    int i0 = blockIdx.x*blockDim.x + threadIdx.x;
    int i1 = blockIdx.y*blockDim.y + threadIdx.y;

    #pragma unroll
    for (int j1 = -KERNEL_RADIUS; j1 < KERNEL_RADIUS; ++j1) {
	if( ((i1+j1)>=0) && ((i1+j1)<imageH)) {
	    out[i0+imageH*i1] += in[i0+imageH*(i1+j1)] * c1[KERNEL_RADIUS+j1];
	};
    };
};


void GPUfilter_y(uint16_t *out, uint16_t *in, int imageW, int imageH){
    int size = imageW*imageH;
    uint16_t *d_out = NULL;
    hipHostRegister(out, size*sizeof(uint16_t), hipHostRegisterMapped);
    hipHostGetDevicePointer((void **)&d_out, (void *)out, 0);
    uint16_t *d_in = NULL;
    hipHostRegister(in, size*sizeof(uint16_t), hipHostRegisterMapped);
    hipHostGetDevicePointer((void **)&d_in, (void *)in, 0);

    dim3 threadsPerBlock(1,512);
    dim3 numBlocks(imageW/threadsPerBlock.x, imageH/threadsPerBlock.y);
    
    filter_y<<<numBlocks,threadsPerBlock>>>(d_out,d_in,imageW,imageH);
    // clean up
    hipHostUnregister(d_out);
    hipHostUnregister(d_in);
};